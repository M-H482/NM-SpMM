#include <algorithm>
#include <cmath>
#include <hip/hip_runtime.h>

#include <iostream>
#include <omp.h>
#include <random>
#include <time.h>
using namespace std;

// printf("%s %d CUDA: %s\n", __FILE__, __LINE__, cudaGetErrorString(cudaGetLastError()));

#define alignN 32
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

void init_data(float* A, float* B, int* D, float* BT, int* DT, float* C, int M, int N, int K, int pruning_M, float sparsity)
{
    /**
     *  A:      col-major
     *  B:      col-major
     *  D:      col-major
     *  BT:     row-major
     *  DT:     row-major
     */

    // generate different seed for random number
    time_t t;
    srand((unsigned)time(&t));
    // srand(1);
    const unsigned int W = K * (1.0f - sparsity);
    const unsigned int pruning_N = pruning_M * (1.0f - sparsity);
    const unsigned int Q = (int)(N / alignN);

    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            C[i + j * M] = 0.0f;
        }
    }

    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < K; ++j) {
            A[i + j * M] = (float)rand() / RAND_MAX;
        }
    }

    for (int i = 0; i < W; ++i) {
        for (int j = 0; j < N; ++j) {
            B[i + j * W] = (float)rand() / RAND_MAX;
            BT[i * N + j] = B[i + j * W];
        }
    }

    int* tmp_index = (int*)malloc(sizeof(int) * pruning_M);
    for (int i = 0; i < pruning_M; ++i)
        tmp_index[i] = i;

    std::mt19937 gen(std::random_device {}());
    for (int j = 0; j < N; j += alignN) {
        int a = (int)(j / alignN);
        for (int k = 0; k < W; k += pruning_N) {

            std::shuffle(tmp_index, tmp_index + pruning_M, gen);
            // std::random_shuffle(tmp_index, tmp_index + pruning_M);
            std::sort(tmp_index, tmp_index + pruning_N);

            for (int u = 0; u < pruning_N; ++u) {
                // for(int iner_j = 0; iner_j < alignN; iner_j++){
                //     D[(k + u) + (j + iner_j) * W] = tmp_index[u] + k / (1.0f - sparsity);
                //     DT[(k + u) * N + (j + iner_j)] = D[(k + u) + (j + iner_j) * W];
                // }

                D[(k + u) + a * W] = tmp_index[u] + k / (1.0f - sparsity);
                // // D[(k + u) + a * W] = tmp_index[u];
                // DT[(k + u) * Q + a] = D[(k + u) + a * W];
                DT[(k + u) * Q + a] = tmp_index[u];
            }
        }
    }
    free(tmp_index);
}

void PreProcessing(int* DT, int W, int Q, int Ns)
{
    int Qs = Ns / alignN;
    // layout transform
    int* buffer = (int*)malloc(sizeof(int) * W * Q);
    for (int j = 0; j < Q; j += Qs) {
        int* p = buffer + j * W;
        for (int row = 0; row < W; row++) {
            for (int col = 0; col < Qs; col++) {
                *p = DT[row * Q + j + col];
                p += 1;
            }
        }
    }
    for (int i = 0; i < W; i++) {
        for (int j = 0; j < Q; j++) {
            DT[i * Q + j] = buffer[i * Q + j];
        }
    }
    free(buffer);
}

template <
    const int Ms,
    const int Ns,
    const int Ks,
    const int Ws,
    const int Mt,
    const int Nt>
__global__ void nmGEMM(float* A, float* B, int* D, float* C, int M, int N, int K, int W)
{
    /*
     *    A, B, D, C: col-major, row-major, row-major, row-major
     */
    const int Qs = (Ns + alignN - 1) / alignN;

    extern __shared__ char smem[];
    float At[Mt], Bt[Nt], Ct[Mt][Nt] = { 0.0f };

    float* As = (float*)smem;
    float* Bs = (float*)(smem + Ks * Ms * sizeof(float));
    int* Ds = (int*)(smem + (Ks * Ms + Ws * Ns) * sizeof(float));

    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int warp_id = tid / warpSize;
    const int lane_id = tid % warpSize;

    const int bi = blockIdx.y * Ms;
    const int bj = blockIdx.x * Ns;

    const int ti = ((lane_id % 16) / 2) * 4;
    const int tj = warp_id * 32 + (lane_id / 16) * 8 + (lane_id % 2) * 4;

    const int THREADS_PER_BLOCK = (Ms / Mt) * (Ns / Nt);

    const int A_THREADS_PER_ROW = Ms / 4;
    const int B_THREADS_PER_ROW = Ns / 4;

    const int A_STRIDES = THREADS_PER_BLOCK / A_THREADS_PER_ROW;
    const int B_STRIDES = THREADS_PER_BLOCK / B_THREADS_PER_ROW;

    int A_BLOCK_ROW_START = tid / A_THREADS_PER_ROW;
    int B_BLOCK_ROW_START = tid / B_THREADS_PER_ROW;

    int A_BLOCK_COL_START = tid % A_THREADS_PER_ROW * 4;
    int B_BLOCK_COL_START = tid % B_THREADS_PER_ROW * 4;

    const int load_D_num = Ws * Qs;

    for (int u = 0, v = 0; u < W; u += Ws, v += Ks) {
        float* A_ptr = A + bi + v * M;
        float* B_ptr = B + bj + u * N;
        int* D_ptr = D + bj / alignN * W + u * Qs;

#pragma unroll
        for (int i = 0; i < Ks; i += A_STRIDES) {
            FETCH_FLOAT4(As[(i + A_BLOCK_ROW_START) * Ms + A_BLOCK_COL_START])
                = FETCH_FLOAT4(A_ptr[(i + A_BLOCK_ROW_START) * M + A_BLOCK_COL_START]);
        }
#pragma unroll
        for (int i = 0; i < Ws; i += B_STRIDES) {
            FETCH_FLOAT4(Bs[(i + B_BLOCK_ROW_START) * Ns + B_BLOCK_COL_START])
                = FETCH_FLOAT4(B_ptr[(i + B_BLOCK_ROW_START) * N + B_BLOCK_COL_START]);
        }
        if (tid < load_D_num) {
            Ds[tid] = D_ptr[tid];
        }

        __syncthreads();
#pragma unroll
        for (int p = 0; p < Ws; p++) {
            FETCH_FLOAT4(At[0]) = FETCH_FLOAT4(As[Ds[p * Qs + tj / alignN] * Ms + ti]);
            FETCH_FLOAT4(At[4]) = FETCH_FLOAT4(As[Ds[p * Qs + tj / alignN] * Ms + ti + 32]);
            FETCH_FLOAT4(Bt[0]) = FETCH_FLOAT4(Bs[p * Ns + tj]);
            FETCH_FLOAT4(Bt[4]) = FETCH_FLOAT4(Bs[p * Ns + tj + 16]);
#pragma unroll
            for (int i = 0; i < Mt; i++) {
#pragma unroll
                for (int j = 0; j < Nt; j++) {
                    Ct[i][j] += At[i] * Bt[j];
                }
            }
        }
        __syncthreads();
    }
#pragma unroll
    for (int i = 0; i < 4; i++) {
        FETCH_FLOAT4(C[(bi + ti + i) * N + (bj + tj + 0)]) = FETCH_FLOAT4(Ct[i][0]);
        FETCH_FLOAT4(C[(bi + ti + i) * N + (bj + tj + 16)]) = FETCH_FLOAT4(Ct[i][4]);
        FETCH_FLOAT4(C[(bi + ti + i + 32) * N + (bj + tj + 0)]) = FETCH_FLOAT4(Ct[i + 4][0]);
        FETCH_FLOAT4(C[(bi + ti + i + 32) * N + (bj + tj + 16)]) = FETCH_FLOAT4(Ct[i + 4][4]);
    }
}

void nmspmm(float* A, float* B, int* D, float* C, int M, int N, int K, int W, float sparsity)
{
    const int Ms = 64;
    const int Ns = 128;
    const int Mt = 8;
    const int Nt = 8;

    dim3 dimBlock(Ns / Nt, Ms / Mt);
    dim3 dimGrid(N / Ns, M / Ms);

    if (fabs(sparsity - 0.75f) < 1e-6 || fabs(sparsity - 0.875f) < 1e-6) {
        printf("Not support! Please use low sparsity version for sparsity %.3f\n", sparsity);
    } else if (fabs(sparsity - 0.5f) < 1e-6) {
        const int Ks = 32;
        const int Ws = 16;
        size_t smem_nbytes = (Ks * Ms + Ws * Ns) * sizeof(float)
            + (Ws * Ns / alignN + Ks) * sizeof(int);
        nmGEMM<Ms, Ns, Ks, Ws, Mt, Nt>
            <<<dimGrid, dimBlock, smem_nbytes>>>(A, B, D, C, M, N, K, W);
    } else if (fabs(sparsity - 0.625f) < 1e-6) {
        const int Ks = 32;
        const int Ws = 12;
        size_t smem_nbytes = (Ks * Ms + Ws * Ns) * sizeof(float)
            + (Ws * Ns / alignN + Ks) * sizeof(int);
        nmGEMM<Ms, Ns, Ks, Ws, Mt, Nt>
            <<<dimGrid, dimBlock, smem_nbytes>>>(A, B, D, C, M, N, K, W);
    }
}

void matmul_on_cpu(float* A, float* B, int* D, float* C, int M, int N, int K, int W)
{
    int num_threads = omp_get_max_threads();
    printf("Using %d threads compute reference on CPU\n", num_threads);
#pragma omp parallel for
    for (int i = 0; i < M; i++) {
        for (int k = 0; k < W; k++) {
            for (int j = 0; j < N; j++) {
                int a = (int)(j / alignN);
                C[i * N + j] += A[i + D[k + a * W] * M] * B[k + j * W];
            }
        }
    }
}

bool allclose(float* A, float* B, int n)
{
    // absolute(a - b) <= (atol + rtol * absolute(b))
    for (int i = 0; i < n; i++) {
        float a = A[i], b = B[i];
        float rtol = 1e-5, atol = 1e-8;
        if (!(fabs(a - b) <= (atol + rtol * fabs(b)))) {
            printf("Error on index %d, (%f, %f)\n", i, a, b);
            return false;
        }
    }
    return true;
}

void trans_inplace(float* a, int m, int n)
{
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            swap(a[i * n + j], a[i + j * m]);
        }
    }
}

int main(int argc, char** argv)
{
    int M = 4096;
    int N = 4096;
    int K = 4096;
    int pruning_M = 32;
    float sparsity = 0.5f;
    int warm_up = 100, iter = 100;

    if (argc == 4) {
        M = atoi(argv[1]);
        N = atoi(argv[2]);
        K = atoi(argv[3]);
    } else if (argc == 6) {
        M = atoi(argv[1]);
        N = atoi(argv[2]);
        K = atoi(argv[3]);
        pruning_M = atoi(argv[4]);
        sparsity = atof(argv[5]);
    } else if (argc == 8) {
        M = atoi(argv[1]);
        N = atoi(argv[2]);
        K = atoi(argv[3]);
        pruning_M = atoi(argv[4]);
        sparsity = atof(argv[5]);
        warm_up = atoi(argv[6]);
        iter = atoi(argv[7]);
    }

    int W = (int)(K * (1.0f - sparsity));
    int Q = (int)(N / alignN);
    int pruning_N = (int)(pruning_M * (1.0f - sparsity));

    printf("M = %d, N = %d, K = %d, pruning_M = %d, sparsity = %f\n", M, N, K, pruning_M, sparsity);
    // ***************** initialize  *******************
    const int A_nBytes = sizeof(float) * M * K;
    const int C_nBytes = sizeof(float) * M * N;

    const int B_nBytes = sizeof(float) * W * N;
    const int D_nBytes = sizeof(int) * W * Q;

    float* hA = (float*)malloc(A_nBytes);
    float* hB = (float*)malloc(B_nBytes);
    float* hB_T = (float*)malloc(B_nBytes);
    float* hC = (float*)malloc(C_nBytes);

    float* hostRef = (float*)malloc(C_nBytes);
    float* deviceRes = (float*)malloc(C_nBytes);

    int* hD = (int*)malloc(D_nBytes);
    int* hD_T = (int*)malloc(D_nBytes);

    init_data(hA, hB, hD, hB_T, hD_T, hC, M, N, K, pruning_M, sparsity);

    int Ns = 128;
    PreProcessing(hD_T, W, Q, Ns);

    float *dA, *dB, *dC;
    int* dD;
    hipMalloc((void**)&dA, A_nBytes);
    hipMalloc((void**)&dB, B_nBytes);
    hipMalloc((void**)&dC, C_nBytes);
    hipMalloc((void**)&dD, D_nBytes);

    hipMemcpy(dA, hA, A_nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB_T, B_nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dC, hC, C_nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dD, hD_T, D_nBytes, hipMemcpyHostToDevice);

    // ***************** result check *******************

    // matmul_on_cpu(hA, hB, hD, hostRef, M, N, K, W);
    // // trans_inplace(hostRef, M, N);
    // nmspmm(dA, dB, dD, dC, M, N, K, W, sparsity);
    // cudaDeviceSynchronize();
    // cudaMemcpy(deviceRes, dC, C_nBytes, cudaMemcpyDeviceToHost);

    // if (allclose(deviceRes, hostRef, M * N)) {
    //     printf("The result is right!\n");
    // } else {
    //     printf("The result is wrong !!!!!!!!!!\n");
    // }

    // ***************** profling.    *******************

    for (int i = 0; i < 3; i++) {
        nmspmm(dA, dB, dD, dC, M, N, K, W, sparsity);
    }

    // ***************** calculate performance *******************
    float milliseconds = 0.0f, tflops = -1.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int i = 0; i < warm_up; i++) {
        nmspmm(dA, dB, dD, dC, M, N, K, W, sparsity);
    }
    hipDeviceSynchronize();
    hipEventRecord(start);

    for (int i = 0; i < iter; i++) {
        nmspmm(dA, dB, dD, dC, M, N, K, W, sparsity);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    milliseconds = milliseconds / iter;

    tflops = (2.0f * M * N * K / 1e12) / (milliseconds / 1e3);
    printf("Time elapsed: %f ms, %f TFLOPS\n", milliseconds, tflops);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
