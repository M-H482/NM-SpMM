#include <algorithm>
#include <cmath>
#include <hip/hip_runtime.h>

#include <iostream>
#include <omp.h>
#include <time.h>
using namespace std;

#define alignN 32

void init_data(float* A, float* B, int* B_idx, float* BT, int* B_idxT, float* C, int M, int N, int K, int pruning_M, float sparsity)
{
    /**
     *  A:      col-major
     *  B:      col-major
     *  B_idx:  col-major
     *  BT:     row-major
     *  B_idxT: row-major
     */

    // generate different seed for random number
    time_t t;
    srand((unsigned)time(&t));
    // srand(1);
    const unsigned int W = K * (1.0f - sparsity);
    const unsigned int pruning_N = pruning_M * (1.0f - sparsity);
    const unsigned int Q = (int)(N / alignN);

    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            C[i + j * M] = 0.0f;
        }
    }

    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < K; ++j) {
            A[i + j * M] = (float)rand() / RAND_MAX;
        }
    }

    for (int i = 0; i < W; ++i) {
        for (int j = 0; j < N; ++j) {
            B[i + j * W] = (float)rand() / RAND_MAX;
            BT[i * N + j] = B[i + j * W];
        }
    }

    int* tmp_index = (int*)malloc(sizeof(int) * pruning_M);
    for (int i = 0; i < pruning_M; ++i)
        tmp_index[i] = i;

    for (int j = 0; j < N; j += alignN) {
        int a = (int)(j / alignN);
        for (int k = 0; k < W; k += pruning_N) {

            std::random_shuffle(tmp_index, tmp_index + pruning_M);
            std::sort(tmp_index, tmp_index + pruning_N);

            for (int u = 0; u < pruning_N; ++u) {
                // for(int iner_j = 0; iner_j < alignN; iner_j++){
                //     B_idx[(k + u) + (j + iner_j) * W] = tmp_index[u] + k / (1.0f - sparsity);
                //     B_idxT[(k + u) * N + (j + iner_j)] = B_idx[(k + u) + (j + iner_j) * W];
                // }

                B_idx[(k + u) + a * W] = tmp_index[u] + k / (1.0f - sparsity);
                // // B_idx[(k + u) + a * W] = tmp_index[u];
                // B_idxT[(k + u) * Q + a] = B_idx[(k + u) + a * W];
                B_idxT[(k + u) * Q + a] = tmp_index[u] + k / (1.0f - sparsity);
            }
        }
    }
    free(tmp_index);
}

template <
    const int Ms, // height of block of C that each thread block calculate
    const int Ns // width of block of C that each thread block load into shared memory
    >
__global__ void nmGEMM(float* A, float* B, int* D, float* C, int M, int N, int K, int W)
{
    /*
     *    A, B, D, C: col-major, row-major, row-major, row-major
     */
    int tid = threadIdx.x;
    int tx = tid % Ns;
    int ty = tid / Ns;

    int i = blockIdx.y * Ms + ty;
    int j = blockIdx.x * Ns + tx;
    float sum = 0.0f;

    const int Q = N / alignN;
    for (int u = 0; u < W; u++) {
        int t = D[u * Q + j / alignN];
        sum += A[i + t * M] * B[u * N + j];
    }
    C[i * N + j] = sum;
}

void nmspmm(float* A, float* B, int* B_idx, float* C, int M, int N, int K, int W, float sparsity)
{
    const int Ms = 32;
    const int Ns = 32;

    dim3 dimBlock(Ms * Ns);
    dim3 dimGrid(N / Ns, M / Ms);

    nmGEMM<Ms, Ns>
        <<<dimGrid, dimBlock>>>(A, B, B_idx, C, M, N, K, W);
}

void matmul_on_cpu(float* A, float* B, int* B_idx, float* C, int M, int N, int K, int W)
{
    int num_threads = omp_get_max_threads();
    printf("Using %d threads compute reference on CPU\n", num_threads);
#pragma omp parallel for
    for (int i = 0; i < M; i++) {
        for (int k = 0; k < W; k++) {
            for (int j = 0; j < N; j++) {
                int a = (int)(j / alignN);
                C[i * N + j] += A[i + B_idx[k + a * W] * M] * B[k + j * W];
            }
        }
    }
}

bool allclose(float* A, float* B, int n)
{
    // absolute(a - b) <= (atol + rtol * absolute(b))
    for (int i = 0; i < n; i++) {
        float a = A[i], b = B[i];
        float rtol = 1e-5, atol = 1e-8;
        if (!(fabs(a - b) <= (atol + rtol * fabs(b)))) {
            printf("Error on index %d, (%f, %f)\n", i, a, b);
            return false;
        }
    }
    return true;
}

void trans_inplace(float* a, int m, int n)
{
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            swap(a[i * n + j], a[i + j * m]);
        }
    }
}

int main(int argc, char** argv)
{
    int M = 4096;
    int N = 4096;
    int K = 4096;
    int pruning_M = 32;
    float sparsity = 0.5f;
    int warm_up = 10, iter = 10;

    if (argc == 4) {
        M = atoi(argv[1]);
        N = atoi(argv[2]);
        K = atoi(argv[3]);
    } else if (argc == 6) {
        M = atoi(argv[1]);
        N = atoi(argv[2]);
        K = atoi(argv[3]);
        pruning_M = atoi(argv[4]);
        sparsity = atof(argv[5]);
    } else if (argc == 8) {
        M = atoi(argv[1]);
        N = atoi(argv[2]);
        K = atoi(argv[3]);
        pruning_M = atoi(argv[4]);
        sparsity = atof(argv[5]);
        warm_up = atoi(argv[6]);
        iter = atoi(argv[7]);
    }

    int W = (int)(K * (1.0f - sparsity));
    int Q = (int)(N / alignN);

    printf("M = %d, N = %d, K = %d, pruning_M = %d, sparsity = %f\n", M, N, K, pruning_M, sparsity);
    // ***************** initialize  *******************
    const int A_nBytes = sizeof(float) * M * K;
    const int C_nBytes = sizeof(float) * M * N;

    const int B_nBytes = sizeof(float) * W * N;
    const int B_idx_nBytes = sizeof(int) * W * Q;

    float* hA = (float*)malloc(A_nBytes);
    float* hB = (float*)malloc(B_nBytes);
    float* hB_T = (float*)malloc(B_nBytes);
    float* hC = (float*)malloc(C_nBytes);

    float* hostRef = (float*)malloc(C_nBytes);
    float* deviceRes = (float*)malloc(C_nBytes);

    int* hB_idx = (int*)malloc(B_idx_nBytes);
    int* hB_T_idx = (int*)malloc(B_idx_nBytes);

    init_data(hA, hB, hB_idx, hB_T, hB_T_idx, hC, M, N, K, pruning_M, sparsity);

    float *dA, *dB, *dC;
    int* dB_idx;
    hipMalloc((void**)&dA, A_nBytes);
    hipMalloc((void**)&dB, B_nBytes);
    hipMalloc((void**)&dC, C_nBytes);
    hipMalloc((void**)&dB_idx, B_idx_nBytes);

    hipMemcpy(dA, hA, A_nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB_T, B_nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dC, hC, C_nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dB_idx, hB_T_idx, B_idx_nBytes, hipMemcpyHostToDevice);

    // ***************** result check *******************

    // matmul_on_cpu(hA, hB, hB_idx, hostRef, M, N, K, W);
    // // trans_inplace(hostRef, M, N);
    // nmspmm(dA, dB, dB_idx, dC, M, N, K, W, sparsity);
    // cudaDeviceSynchronize();
    // cudaMemcpy(deviceRes, dC, C_nBytes, cudaMemcpyDeviceToHost);

    // if (allclose(deviceRes, hostRef, M * N)) {
    //     printf("The result is right!\n");
    // } else {
    //     printf("The result is wrong !!!!!!!!!!\n");
    // }

    // ***************** profling.    *******************

    for (int i = 0; i < 3; i++) {
        nmspmm(dA, dB, dB_idx, dC, M, N, K, W, sparsity);
    }

    // ***************** calculate performance *******************
    float milliseconds = 0.0f, tflops = -1.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int i = 0; i < warm_up; i++) {
        nmspmm(dA, dB, dB_idx, dC, M, N, K, W, sparsity);
    }
    hipDeviceSynchronize();
    hipEventRecord(start);

    for (int i = 0; i < iter; i++) {
        nmspmm(dA, dB, dB_idx, dC, M, N, K, W, sparsity);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    milliseconds = milliseconds / iter;

    tflops = (2.0f * M * N * K / 1e12) / (milliseconds / 1e3);
    printf("Time elapsed: %f ms, %f TFLOPS\n", milliseconds, tflops);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
