#include <algorithm>
#include <cmath>
#include <hip/hip_runtime.h>

#include <iostream>
#include <omp.h>
#include <random>
#include <time.h>
using namespace std;

// printf("%s %d CUDA: %s\n", __FILE__, __LINE__, cudaGetErrorString(cudaGetLastError()));

#define alignN 32
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

#define CP_ASYNC_CA(dst, src, Bytes) \
    asm volatile("cp.async.ca.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(Bytes))

#define CP_ASYNC_CA_Guard(dst, src, Bytes, guard)                                \
    asm volatile(                                                                \
        "{.reg .pred p;\n"                                                       \
        " setp.ne.b32 p, %3, 0;\n"                                               \
        " @p cp.async.ca.shared.global.L2::128B [%0], [%1], %2; }\n" ::"r"(dst), \
        "l"(src), "n"(Bytes), "r"((int)(guard)))

#define CP_ASYNC_CG(dst, src, Bytes) \
    asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(Bytes))

#define CP_ASYNC_CG_Guard(dst, src, Bytes, guard)                                \
    asm volatile(                                                                \
        "{.reg .pred p;\n"                                                       \
        " setp.ne.b32 p, %3, 0;\n"                                               \
        " @p cp.async.cg.shared.global.L2::128B [%0], [%1], %2; }\n" ::"r"(dst), \
        "l"(src), "n"(Bytes), "r"((int)(guard)))

#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)
#define CP_ASYNC_WAIT_GROUP(N) asm volatile("cp.async.wait_group %0;\n" ::"n"(N))
#define CP_ASYNC_WAIT_ALL() asm volatile("cp.async.wait_all;\n" ::)

__device__ inline void atomicAddFloat4(float* address, float* val)
{
    atomicAdd(address + 0, *(val + 0));
    atomicAdd(address + 1, *(val + 1));
    atomicAdd(address + 2, *(val + 2));
    atomicAdd(address + 3, *(val + 3));
}

void init_data(float* A, float* B, int* D, float* BT, int* DT, float* C, int M, int N, int K, int pruning_M, float sparsity)
{
    /**
     *  A:      col-major
     *  B:      col-major
     *  D:      col-major
     *  BT:     row-major
     *  DT:     row-major
     */

    // generate different seed for random number
    time_t t;
    srand((unsigned)time(&t));
    // srand(1);
    const unsigned int W = K * (1.0f - sparsity);
    const unsigned int pruning_N = pruning_M * (1.0f - sparsity);
    const unsigned int Q = (int)(N / alignN);

    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            C[i + j * M] = 0.0f;
        }
    }

    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < K; ++j) {
            A[i + j * M] = (float)rand() / RAND_MAX;
        }
    }

    for (int i = 0; i < W; ++i) {
        for (int j = 0; j < N; ++j) {
            B[i + j * W] = (float)rand() / RAND_MAX;
            BT[i * N + j] = B[i + j * W];
        }
    }

    int* tmp_index = (int*)malloc(sizeof(int) * pruning_M);
    for (int i = 0; i < pruning_M; ++i)
        tmp_index[i] = i;

    std::mt19937 gen(std::random_device {}());
    for (int j = 0; j < N; j += alignN) {
        int a = (int)(j / alignN);
        for (int k = 0; k < W; k += pruning_N) {

            std::shuffle(tmp_index, tmp_index + pruning_M, gen);
            // std::random_shuffle(tmp_index, tmp_index + pruning_M);
            std::sort(tmp_index, tmp_index + pruning_N);

            for (int u = 0; u < pruning_N; ++u) {
                // for(int iner_j = 0; iner_j < alignN; iner_j++){
                //     D[(k + u) + (j + iner_j) * W] = tmp_index[u] + k / (1.0f - sparsity);
                //     DT[(k + u) * N + (j + iner_j)] = D[(k + u) + (j + iner_j) * W];
                // }

                D[(k + u) + a * W] = tmp_index[u] + k / (1.0f - sparsity);
                // // D[(k + u) + a * W] = tmp_index[u];
                // DT[(k + u) * Q + a] = D[(k + u) + a * W];
                DT[(k + u) * Q + a] = tmp_index[u];
            }
        }
    }
    free(tmp_index);
}

void transIndex(int* DT, int W, int N, int Q, int& pruning_N, int& pruning_M)
{
    for (int j = 0; j < N; j += alignN) {
        int a = (int)(j / alignN);
        for (int k = pruning_N; k < W; k += pruning_N * 2) {
            for (int u = 0; u < pruning_N; ++u) {
                DT[(k + u) * Q + a] += pruning_M;
            }
        }
    }
    pruning_M *= 2;
    pruning_N *= 2;
}

void PreProcessing(int* column_info, int pruning_M, int pruning_N, int* DT, int W, int Q, int N, int Ns)
{
    // re indexing
    int Qs = Ns / alignN;
    int* column_info_ptr = column_info;

    // indices reordering & get colinfo
    for (int j = 0; j < Q; j += Qs) {
        for (int i = 0; i < W; i += pruning_N) {
            for (int x = 0; x < pruning_M; x++)
                column_info_ptr[x] = -1;

            int bucket[pruning_M];
            for (int x = 0; x < pruning_M; x++)
                bucket[x] = 0;

            for (int x = 0; x < pruning_N; x++) {
                for (int y = 0; y < Qs; y++) {
                    int v = DT[(i + x) * Q + (j + y)];
                    bucket[v] += 1;
                }
            }

            int map[pruning_M];
            for (int x = 0, y = 0; x < pruning_M; x++) {
                int v = bucket[x];
                if (v > 0) {
                    column_info_ptr[y] = x;
                    map[x] = y;
                    y++;
                }
            }

            for (int x = 0; x < pruning_N; x++) {
                for (int y = 0; y < Qs; y++) {
                    int v = DT[(i + x) * Q + (j + y)];
                    DT[(i + x) * Q + (j + y)] = map[v];
                }
            }

            column_info_ptr = column_info_ptr + pruning_M;
        }
    }

    // layout transform
    int* buffer = (int*)malloc(sizeof(int) * W * Q);
    for (int j = 0; j < Q; j += Qs) {
        int* p = buffer + j * W;
        for (int row = 0; row < W; row++) {
            for (int col = 0; col < Qs; col++) {
                *p = DT[row * Q + j + col];
                p += 1;
            }
        }
    }
    for (int i = 0; i < W; i++) {
        for (int j = 0; j < Q; j++) {
            DT[i * Q + j] = buffer[i * Q + j];
        }
    }
    free(buffer);
}

template <
    const int Ms,
    const int Ns,
    const int Ks,
    const int Ws,
    const int Mt,
    const int Nt>
__global__ void nmGEMM(float* A, float* B, int* D, int* column_info, float* C, int M, int N, int K, int W)
{
    /*
     *    A, B, D, C: col-major, row-major, row-major, row-major
     */
    int SPLIT_K = gridDim.z;
    int bz = blockIdx.z;
    const int Qs = (Ns + alignN - 1) / alignN;

    int K_LEN, SPLIT_K_OFFSET_A, SPLIT_K_OFFSET_B, SPLIT_K_OFFSET_D, SPLIT_K_OFFSET_I;
    int iter_num = W / Ws;
    if (iter_num % SPLIT_K == 0) {
        K_LEN = W / SPLIT_K;
        SPLIT_K_OFFSET_A = (iter_num / SPLIT_K) * Ks * M * bz;
        SPLIT_K_OFFSET_I = (iter_num / SPLIT_K) * Ks * 1 * bz;
        SPLIT_K_OFFSET_B = K_LEN * N * bz;
        SPLIT_K_OFFSET_D = K_LEN * Qs * bz;
    } else {
        // 20个任务均匀分为8份：3 3 3 3 2 2 2 2
        int p = iter_num / SPLIT_K;
        int q = iter_num % SPLIT_K;
        int offset = (bz < q) ? (bz * p + bz) : (bz * p + q);

        K_LEN = ((bz < q) ? (p + 1) : p) * Ws;
        SPLIT_K_OFFSET_A = offset * Ks * M;
        SPLIT_K_OFFSET_I = offset * Ks * 1;
        SPLIT_K_OFFSET_B = offset * Ws * N;
        SPLIT_K_OFFSET_D = offset * Ws * Qs;
    }

    extern __shared__ char smem[];
    float At[2][Mt], Bt[2][Nt], Ct[Mt][Nt] = { 0.0f };

    float* As_write_ptr = (float*)smem; // [Ks][Ms]
    float* As_read_ptr = As_write_ptr + Ks * Ms;

    float* Bs_write_ptr = (float*)(smem + 2 * Ks * Ms * sizeof(float));
    float* Bs_read_ptr = Bs_write_ptr + Ws * Ns; // [Ws][Ns]

    int* Ds_write_ptr = (int*)(smem + 2 * (Ks * Ms + Ws * Ns) * sizeof(float));
    int* Ds_read_ptr = Ds_write_ptr + Ws * Qs; // [Ws][Qs]

    int* col_info_write_ptr = (int*)(smem + 2 * (Ks * Ms + Ws * Ns) * sizeof(float) + 2 * Ws * Qs * sizeof(int));
    int* col_info_read_ptr = col_info_write_ptr + Ks; // [Ks]

    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int warp_id = tid / warpSize;
    const int lane_id = tid % warpSize;

    const int bi = blockIdx.y * Ms;
    const int bj = blockIdx.x * Ns;

    const int ti = ((lane_id % 16) / 2) * 4;
    const int tj = warp_id * 32 + (lane_id / 16) * 8 + (lane_id % 2) * 4;

    const int THREADS_PER_BLOCK = (Ms / Mt) * (Ns / Nt);

    const int A_THREADS_PER_ROW = Ms / 4;
    const int B_THREADS_PER_ROW = Ns / 4;

    const int A_STRIDES = THREADS_PER_BLOCK / A_THREADS_PER_ROW;
    const int B_STRIDES = THREADS_PER_BLOCK / B_THREADS_PER_ROW;

    int A_BLOCK_ROW_START = tid / A_THREADS_PER_ROW;
    int B_BLOCK_ROW_START = tid / B_THREADS_PER_ROW;

    int A_BLOCK_COL_START = tid % A_THREADS_PER_ROW * 4;
    int B_BLOCK_COL_START = tid % B_THREADS_PER_ROW * 4;

    float* A_ptr = A + bi + SPLIT_K_OFFSET_A;
    float* B_ptr = B + bj + SPLIT_K_OFFSET_B;
    int* D_ptr = D + bj / alignN * W + SPLIT_K_OFFSET_D;
    int* column_info_ptr = column_info + blockIdx.x * (W / Ws) * Ks + SPLIT_K_OFFSET_I;

    int idx[Ws], col[Ks];
    const int load_D_num = Ws * Qs;

    if (tid < Ks) {
        col_info_write_ptr[tid] = column_info_ptr[tid + Ks];
        col_info_read_ptr[tid] = column_info_ptr[tid];
    }
    __syncthreads();

#pragma unroll
    for (int i = 0; i < Ks; i += A_STRIDES) {
        if (col_info_read_ptr[i + A_BLOCK_ROW_START] != -1) {
            FETCH_FLOAT4(As_write_ptr[(i + A_BLOCK_ROW_START) * Ms + A_BLOCK_COL_START])
                = FETCH_FLOAT4(A_ptr[col_info_read_ptr[i + A_BLOCK_ROW_START] * M + A_BLOCK_COL_START]);
        }
    }
#pragma unroll
    for (int i = 0; i < Ws; i += B_STRIDES) {
        FETCH_FLOAT4(Bs_write_ptr[(i + B_BLOCK_ROW_START) * Ns + B_BLOCK_COL_START])
            = FETCH_FLOAT4(B_ptr[(i + B_BLOCK_ROW_START) * N + B_BLOCK_COL_START]);
    }
    if (tid < load_D_num) {
        Ds_write_ptr[tid] = D_ptr[tid];
    }

    __syncthreads();
#pragma unroll
    for (int p = 0; p < Ws; p++) {
        idx[p] = Ds_write_ptr[p * Qs + tj / alignN];
    }
#pragma unroll
    for (int i = 0; i < Ks; i += A_STRIDES) {
        col[i] = col_info_write_ptr[i + A_BLOCK_ROW_START];
    }

    FETCH_FLOAT4(Bt[0][0]) = FETCH_FLOAT4(Bs_write_ptr[0 * Ns + tj]);
    FETCH_FLOAT4(Bt[0][4]) = FETCH_FLOAT4(Bs_write_ptr[0 * Ns + tj + 16]);
    FETCH_FLOAT4(At[0][0]) = FETCH_FLOAT4(As_write_ptr[idx[0] * Ms + ti]);
    FETCH_FLOAT4(At[0][4]) = FETCH_FLOAT4(As_write_ptr[idx[0] * Ms + ti + 32]);

    for (int u = Ws, v = Ks; u < K_LEN; u += Ws, v += Ks) {

        A_ptr = A + bi + SPLIT_K_OFFSET_A + v * M;
        B_ptr = B + bj + SPLIT_K_OFFSET_B + u * N;
        D_ptr = D + bj / alignN * W + SPLIT_K_OFFSET_D + u * Qs;
        column_info_ptr = column_info + blockIdx.x * (W / Ws) * Ks + SPLIT_K_OFFSET_I + v + Ks;

        {
            float* t;
            t = As_read_ptr, As_read_ptr = As_write_ptr, As_write_ptr = t;
            t = Bs_read_ptr, Bs_read_ptr = Bs_write_ptr, Bs_write_ptr = t;
        }
        {
            int* t;
            t = Ds_read_ptr, Ds_read_ptr = Ds_write_ptr, Ds_write_ptr = t;
            t = col_info_read_ptr, col_info_read_ptr = col_info_write_ptr, col_info_write_ptr = t;
        }

        {
            uint32_t addr = __cvta_generic_to_shared(&col_info_write_ptr[tid]);
            CP_ASYNC_CA_Guard(addr, column_info_ptr + tid, 4, tid < Ks);
        }

        {
            uint32_t addr = __cvta_generic_to_shared(&Ds_write_ptr[tid]);
            CP_ASYNC_CA_Guard(addr, D_ptr + tid, 4, tid < load_D_num);
        }

#pragma unroll
        for (int i = 0; i < Ks; i += A_STRIDES) {
            uint32_t addr = __cvta_generic_to_shared(&As_write_ptr[(i + A_BLOCK_ROW_START) * Ms + A_BLOCK_COL_START]);
            CP_ASYNC_CG_Guard(addr, &A_ptr[col[i] * M + A_BLOCK_COL_START], 16, col[i] != -1);
        }
#pragma unroll
        for (int i = 0; i < Ws; i += B_STRIDES) {
            uint32_t addr = __cvta_generic_to_shared(&Bs_write_ptr[(i + B_BLOCK_ROW_START) * Ns + B_BLOCK_COL_START]);
            CP_ASYNC_CG(addr, &B_ptr[(i + B_BLOCK_ROW_START) * N + B_BLOCK_COL_START], 16);
        }

        CP_ASYNC_COMMIT_GROUP();

#pragma unroll
        for (int p = 0; p < Ws - 1; p += 1) {
            FETCH_FLOAT4(Bt[(p + 1) % 2][0]) = FETCH_FLOAT4(Bs_read_ptr[(p + 1) * Ns + tj]);
            FETCH_FLOAT4(Bt[(p + 1) % 2][4]) = FETCH_FLOAT4(Bs_read_ptr[(p + 1) * Ns + tj + 16]);
            FETCH_FLOAT4(At[(p + 1) % 2][0]) = FETCH_FLOAT4(As_read_ptr[idx[p + 1] * Ms + ti]);
            FETCH_FLOAT4(At[(p + 1) % 2][4]) = FETCH_FLOAT4(As_read_ptr[idx[p + 1] * Ms + ti + 32]);
#pragma unroll
            for (int i = 0; i < Mt; i++) {
                if (i % 2) {
#pragma unroll
                    for (int j = Nt - 1; j >= 0; j--) {
                        Ct[i][j] += At[p % 2][i] * Bt[p % 2][j];
                    }
                } else {
#pragma unroll
                    for (int j = 0; j < Nt; j++) {
                        Ct[i][j] += At[p % 2][i] * Bt[p % 2][j];
                    }
                }
            }
        }
        CP_ASYNC_WAIT_ALL();
        __syncthreads();

#pragma unroll
        for (int p = 0; p < Ws; p++) {
            idx[p] = Ds_write_ptr[p * Qs + tj / alignN];
        }
#pragma unroll
        for (int i = 0; i < Ks; i += A_STRIDES) {
            col[i] = col_info_write_ptr[i + A_BLOCK_ROW_START];
        }

        FETCH_FLOAT4(Bt[0][0]) = FETCH_FLOAT4(Bs_write_ptr[0 * Ns + tj]);
        FETCH_FLOAT4(Bt[0][4]) = FETCH_FLOAT4(Bs_write_ptr[0 * Ns + tj + 16]);
        FETCH_FLOAT4(At[0][0]) = FETCH_FLOAT4(As_write_ptr[idx[0] * Ms + ti]);
        FETCH_FLOAT4(At[0][4]) = FETCH_FLOAT4(As_write_ptr[idx[0] * Ms + ti + 32]);

#pragma unroll
        for (int i = 0; i < Mt; i++) {
            if (i % 2) {
#pragma unroll
                for (int j = Nt - 1; j >= 0; j--) {
                    Ct[i][j] += At[1][i] * Bt[1][j];
                }
            } else {
#pragma unroll
                for (int j = 0; j < Nt; j++) {
                    Ct[i][j] += At[1][i] * Bt[1][j];
                }
            }
        }
    }

    {
        float* t;
        t = As_read_ptr, As_read_ptr = As_write_ptr, As_write_ptr = t;
        t = Bs_read_ptr, Bs_read_ptr = Bs_write_ptr, Bs_write_ptr = t;
    }
    {
        int* t;
        t = Ds_read_ptr, Ds_read_ptr = Ds_write_ptr, Ds_write_ptr = t;
        t = col_info_read_ptr, col_info_read_ptr = col_info_write_ptr, col_info_write_ptr = t;
    }

#pragma unroll
    for (int p = 0; p < Ws - 1; p++) {
        FETCH_FLOAT4(Bt[(p + 1) % 2][0]) = FETCH_FLOAT4(Bs_read_ptr[(p + 1) * Ns + tj]);
        FETCH_FLOAT4(Bt[(p + 1) % 2][4]) = FETCH_FLOAT4(Bs_read_ptr[(p + 1) * Ns + tj + 16]);
        FETCH_FLOAT4(At[(p + 1) % 2][0]) = FETCH_FLOAT4(As_read_ptr[idx[p + 1] * Ms + ti]);
        FETCH_FLOAT4(At[(p + 1) % 2][4]) = FETCH_FLOAT4(As_read_ptr[idx[p + 1] * Ms + ti + 32]);
#pragma unroll
        for (int i = 0; i < Mt; i++) {
            if (i % 2) {
#pragma unroll
                for (int j = Nt - 1; j >= 0; j--) {
                    Ct[i][j] += At[p % 2][i] * Bt[p % 2][j];
                }
            } else {
#pragma unroll
                for (int j = 0; j < Nt; j++) {
                    Ct[i][j] += At[p % 2][i] * Bt[p % 2][j];
                }
            }
        }
    }
#pragma unroll
    for (int i = 0; i < Mt; i++) {
        if (i % 2) {
#pragma unroll
            for (int j = Nt - 1; j >= 0; j--) {
                Ct[i][j] += At[1][i] * Bt[1][j];
            }
        } else {
#pragma unroll
            for (int j = 0; j < Nt; j++) {
                Ct[i][j] += At[1][i] * Bt[1][j];
            }
        }
    }

    if (SPLIT_K > 1) {
#pragma unroll
        for (int i = 0; i < 4; i++) {
            atomicAddFloat4(&C[(bi + ti + i) * N + (bj + tj + 0)], &Ct[i][0]);
            atomicAddFloat4(&C[(bi + ti + i) * N + (bj + tj + 16)], &Ct[i][4]);
            atomicAddFloat4(&C[(bi + ti + i + 32) * N + (bj + tj + 0)], &Ct[i + 4][0]);
            atomicAddFloat4(&C[(bi + ti + i + 32) * N + (bj + tj + 16)], &Ct[i + 4][4]);
        }
    } else {
#pragma unroll
        for (int i = 0; i < 4; i++) {
            FETCH_FLOAT4(C[(bi + ti + i) * N + (bj + tj + 0)]) = FETCH_FLOAT4(Ct[i][0]);
            FETCH_FLOAT4(C[(bi + ti + i) * N + (bj + tj + 16)]) = FETCH_FLOAT4(Ct[i][4]);
            FETCH_FLOAT4(C[(bi + ti + i + 32) * N + (bj + tj + 0)]) = FETCH_FLOAT4(Ct[i + 4][0]);
            FETCH_FLOAT4(C[(bi + ti + i + 32) * N + (bj + tj + 16)]) = FETCH_FLOAT4(Ct[i + 4][4]);
        }
    }
}

void nmspmm(float* A, float* B, int* D, int* column_info, float* C, int M, int N, int K, int W, float sparsity, int SPLIT_K)
{
    const int Ms = 64;
    const int Ns = 128;
    const int Mt = 8;
    const int Nt = 8;

    dim3 dimBlock(Ns / Nt, Ms / Mt);
    dim3 dimGrid(N / Ns, M / Ms, SPLIT_K);

    if (fabs(sparsity - 0.5f) < 1e-6 || fabs(sparsity - 0.625f) < 1e-6) {
        printf("Not support! Please use low sparsity version for sparsity %.2f\n", sparsity);
    } else if (fabs(sparsity - 0.75f) < 1e-6) {
        const int Ks = 64;
        const int Ws = 16;
        size_t smem_nbytes = 2 * (Ks * Ms + Ws * Ns) * sizeof(float)
            + 2 * (Ws * Ns / alignN + Ks) * sizeof(int);
        hipFuncSetAttribute(reinterpret_cast<const void*>(nmGEMM<Ms, Ns, Ks, Ws, Mt, Nt>), hipFuncAttributeMaxDynamicSharedMemorySize, smem_nbytes);
        nmGEMM<Ms, Ns, Ks, Ws, Mt, Nt>
            <<<dimGrid, dimBlock, smem_nbytes>>>(A, B, D, column_info, C, M, N, K, W);
    } else if (fabs(sparsity - 0.875f) < 1e-6) {
        const int Ks = 64;
        const int Ws = 8;
        size_t smem_nbytes = 2 * (Ks * Ms + Ws * Ns) * sizeof(float)
            + 2 * (Ws * Ns / alignN + Ks) * sizeof(int);
        hipFuncSetAttribute(reinterpret_cast<const void*>(nmGEMM<Ms, Ns, Ks, Ws, Mt, Nt>), hipFuncAttributeMaxDynamicSharedMemorySize, smem_nbytes);
        nmGEMM<Ms, Ns, Ks, Ws, Mt, Nt>
            <<<dimGrid, dimBlock, smem_nbytes>>>(A, B, D, column_info, C, M, N, K, W);
    }
}

void matmul_on_cpu(float* A, float* B, int* D, float* C, int M, int N, int K, int W)
{
    int num_threads = omp_get_max_threads();
    printf("Using %d threads compute reference on CPU\n", num_threads);
#pragma omp parallel for
    for (int i = 0; i < M; i++) {
        for (int k = 0; k < W; k++) {
            for (int j = 0; j < N; j++) {
                int a = (int)(j / alignN);
                C[i * N + j] += A[i + D[k + a * W] * M] * B[k + j * W];
            }
        }
    }
}

bool allclose(float* A, float* B, int n)
{
    // absolute(a - b) <= (atol + rtol * absolute(b))
    for (int i = 0; i < n; i++) {
        float a = A[i], b = B[i];
        float rtol = 1e-5, atol = 1e-8;
        if (!(fabs(a - b) <= (atol + rtol * fabs(b)))) {
            printf("Error on index %d, (%f, %f)\n", i, a, b);
            return false;
        }
    }
    return true;
}

void trans_inplace(float* a, int m, int n)
{
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            swap(a[i * n + j], a[i + j * m]);
        }
    }
}

int main(int argc, char** argv)
{
    int M = 4096;
    int N = 4096;
    int K = 4096;
    int pruning_M = 32;
    float sparsity = 0.5f;
    int SPLIT_K = 1;
    int warm_up = 100, iter = 100;

    if (argc == 4) {
        M = atoi(argv[1]);
        N = atoi(argv[2]);
        K = atoi(argv[3]);
    } else if (argc == 6) {
        M = atoi(argv[1]);
        N = atoi(argv[2]);
        K = atoi(argv[3]);
        pruning_M = atoi(argv[4]);
        sparsity = atof(argv[5]);
    } else if (argc == 7) {
        M = atoi(argv[1]);
        N = atoi(argv[2]);
        K = atoi(argv[3]);
        pruning_M = atoi(argv[4]);
        sparsity = atof(argv[5]);
        SPLIT_K = atoi(argv[6]);
    } else if (argc == 9) {
        M = atoi(argv[1]);
        N = atoi(argv[2]);
        K = atoi(argv[3]);
        pruning_M = atoi(argv[4]);
        sparsity = atof(argv[5]);
        SPLIT_K = atoi(argv[6]);
        warm_up = atoi(argv[7]);
        iter = atoi(argv[8]);
    }

    int W = (int)(K * (1.0f - sparsity));
    int Q = (int)(N / alignN);
    int pruning_N = (int)(pruning_M * (1.0f - sparsity));

    printf("M = %d, N = %d, K = %d, pruning_M = %d, sparsity = %f, SPLIT_K = %d\n", M, N, K, pruning_M, sparsity, SPLIT_K);
    // ***************** initialize  *******************
    const int A_nBytes = sizeof(float) * M * K;
    const int C_nBytes = sizeof(float) * M * N;

    const int B_nBytes = sizeof(float) * W * N;
    const int D_nBytes = sizeof(int) * W * Q;

    float* hA = (float*)malloc(A_nBytes);
    float* hB = (float*)malloc(B_nBytes);
    float* hB_T = (float*)malloc(B_nBytes);
    float* hC = (float*)malloc(C_nBytes);

    float* hostRef = (float*)malloc(C_nBytes);
    float* deviceRes = (float*)malloc(C_nBytes);

    int* hD = (int*)malloc(D_nBytes);
    int* hD_T = (int*)malloc(D_nBytes);

    init_data(hA, hB, hD, hB_T, hD_T, hC, M, N, K, pruning_M, sparsity);
    if (pruning_M == 32) {
        transIndex(hD_T, W, N, Q, pruning_N, pruning_M);
    }

    int Ns = 128;
    int column_nBytes = ((W / pruning_N) * (N / Ns) + 1) * (pruning_M) * sizeof(int);
    int* column_info = (int*)malloc(column_nBytes);

    PreProcessing(column_info, pruning_M, pruning_N, hD_T, W, Q, N, Ns);
    int cnt = 0;
    for (int i = 0; i < (W / pruning_N) * (N / Ns) * (pruning_M); i++) {
        if (column_info[i] == -1) {
            cnt++;
        }
    }
    printf("Reduce %f\n", 1.0f * cnt / ((W / pruning_N) * (N / Ns) * (pruning_M)));

    float *dA, *dB, *dC;
    int *dD, *dcolumn_info;
    hipMalloc((void**)&dA, A_nBytes);
    hipMalloc((void**)&dB, B_nBytes);
    hipMalloc((void**)&dC, C_nBytes);
    hipMalloc((void**)&dD, D_nBytes);
    hipMalloc((void**)&dcolumn_info, column_nBytes);

    hipMemcpy(dA, hA, A_nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB_T, B_nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dC, hC, C_nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dD, hD_T, D_nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dcolumn_info, column_info, column_nBytes, hipMemcpyHostToDevice);

    // ***************** result check *******************

    // matmul_on_cpu(hA, hB, hD, hostRef, M, N, K, W);
    // // trans_inplace(hostRef, M, N);
    // nmspmm(dA, dB, dD, dcolumn_info, dC, M, N, K, W, sparsity, SPLIT_K);
    // cudaDeviceSynchronize();
    // cudaMemcpy(deviceRes, dC, C_nBytes, cudaMemcpyDeviceToHost);

    // if (allclose(deviceRes, hostRef, M * N)) {
    //     printf("The result is right!\n");
    // } else {
    //     printf("The result is wrong !!!!!!!!!!\n");
    //     return 0;
    // }

    // ***************** profling.    *******************

    for (int i = 0; i < 3; i++) {
        nmspmm(dA, dB, dD, dcolumn_info, dC, M, N, K, W, sparsity, SPLIT_K);
    }

    // ***************** calculate performance *******************
    float milliseconds = 0.0f, tflops = -1.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int i = 0; i < warm_up; i++) {
        nmspmm(dA, dB, dD, dcolumn_info, dC, M, N, K, W, sparsity, SPLIT_K);
    }
    hipDeviceSynchronize();
    hipEventRecord(start);

    for (int i = 0; i < iter; i++) {
        nmspmm(dA, dB, dD, dcolumn_info, dC, M, N, K, W, sparsity, SPLIT_K);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    milliseconds = milliseconds / iter;

    tflops = (2.0f * M * N * K / 1e12) / (milliseconds / 1e3);
    printf("Time elapsed: %f ms, %f TFLOPS\n", milliseconds, tflops);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
